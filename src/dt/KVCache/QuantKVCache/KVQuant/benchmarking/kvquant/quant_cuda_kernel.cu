#include "hip/hip_runtime.h"
#include <torch/all.h>
#include <torch/python.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

// half-tensor
#include <c10/cuda/CUDAStream.h>
#include <ATen/cuda/CUDATensorMethods.cuh>

// Thrust
#include <thrust/iterator/counting_iterator.h>
#include <thrust/copy.h>
#include <thrust/functional.h>
#include <thrust/device_vector.h>

// atomicAdd for double-precision floating-point numbers on hardware with
// compute capability < 6.0 from:
// https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#atomic-functions
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 600
__device__ double atomicAdd(
    double* address,
    double val
) {
  unsigned long long int* address_as_ull = (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;

  do {
    assumed = old;
    old = atomicCAS(
      address_as_ull,
      assumed,
      __double_as_longlong(val + __longlong_as_double(assumed))
    );

  // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
  } while (assumed != old);

  return __longlong_as_double(old);
}
#endif

const int BLOCKWIDTH  = 128;
const int BLOCKHEIGHT4 =  16;

const int PBLOCKWIDTH  = 32;
const int PBLOCKHEIGHT4 =  4;

__device__ inline unsigned int as_unsigned(int i) {
  return *reinterpret_cast<unsigned int*>(&i);
}

__device__ inline int as_int(int i) {
  return *reinterpret_cast<int*>(&i);
}

__global__ void VecQuant4AppendVecKSparse(
           int* __restrict__ mat,
  const  float* __restrict__ lookup_table,
  const  float* __restrict__ newvec,
  int*  __restrict__ mask,
  int*  __restrict__ outlier_count,
  int*  __restrict__ outlier_count_per_block,
  const  float* __restrict__ outlier_threshold_lower,
  const  float* __restrict__ outlier_threshold_upper,
  int width,
  int fullwidth
);

__global__ void VecQuant4AppendVecKSparse2(
  const  float* __restrict__ newvec,
  float* __restrict__ zeropoint,
  int*  __restrict__ mask,
  int*  __restrict__ outlier_count_per_block,
  int*  __restrict__ dst_indices,
  float*  __restrict__ dst_values,
  int num_blocks
);

__global__ void VecQuant4AppendVecVSparse(
           int* __restrict__ mat,
         float* __restrict__ lookup_table,
  const  float* __restrict__ newvec,
  int* __restrict__ mask,
  int* __restrict__ outlier_count,
  int*  __restrict__ outlier_count_per_block,
  float outlier_threshold_lower,
  float outlier_threshold_upper,
  int height,
  int fullheight,
  int width,
  int numheads
);

__global__ void VecQuant4AppendVecVSparse2(
  const  float* __restrict__ newvec,
  float zeropoint,
  int*  __restrict__ mask,
  int*  __restrict__ outlier_count_per_block,
  int*  __restrict__ dst_indices,
  float*  __restrict__ dst_values,
  int num_blocks
);

template <typename scalar_t>
__global__ void SPMV_ATOMIC_CSR_ROPE_BALANCED(
    const       int* __restrict__ rows,
    const       int* __restrict__ cols,
    const       int* __restrict__ startrows,
    const  scalar_t* __restrict__ mat,
    const  scalar_t* __restrict__ vec,
	         scalar_t* __restrict__ mul,
    const  int num_rows,
    int numheads,
    int seqlen,
    int headdim,
    int num_threads,
    int nnz,
    float rope_theta,
    int pos_offset
);

template <typename scalar_t>
__global__ void SPMV_ATOMIC_CSC_BALANCED(
    const       int* __restrict__ rows,
    const       int* __restrict__ cols,
    const       int* __restrict__ startcols,
    const  scalar_t* __restrict__ mat,
    const  scalar_t* __restrict__ vec,
	         scalar_t* __restrict__ mul,
    const  int num_cols,
    int numheads,
    int seqlen,
    int headdim,
    int num_threads,
    int nnz
);

__global__ void VecQuant4MatMulKernelNUQPerChannelTransposedMHABatchedFused(
    const  float* __restrict__ vec,
    const    int* __restrict__ mat,
           float* __restrict__ mul,
    const  float* __restrict__ lookup_table,
    const  float* __restrict__ scalingfactor,
    const  float* __restrict__ zeropoint,
    int height,
    int width,
    int fullheight,
    int headdim,
    int numheads,
    int batch_size
);

__global__ void VecQuant4MatMulKernelNUQPerChannelTransposedRopeMHABatchedFused(
    const  float* __restrict__ vec,
    const    int* __restrict__ mat,
           float* __restrict__ mul,
    const  float* __restrict__ lookup_table,
    const  float* __restrict__ scalingfactor,
    const  float* __restrict__ zeropoint,
    int height,
    int width,
    int fullwidth,
    int headdim,
    int numheads,
    int batch_size,
    float rope_theta,
    int pos_offset
);

std::vector<torch::Tensor> vecquant4appendvecKsparse_cuda(
  torch::Tensor mat,
  torch::Tensor lookup_table,
  torch::Tensor newvec,
  torch::Tensor zeropoint,
  torch::Tensor row,
  torch::Tensor col,
  torch::Tensor val,
  torch::Tensor start_rows,
  torch::Tensor outlier_threshold_lower,
  torch::Tensor outlier_threshold_upper,
  int kcachelen
) {

  // mat - kvcache - (num_heads, head_dim, kseqlen)
  int numheads = mat.size(0);
  int height = mat.size(1); // headdim
  int width = kcachelen; // sequence length
  int fullwidth = mat.size(2); // max sequence length

  // lookup table - (num_heads, head_dim, 16)
  int numheads2 = lookup_table.size(0);
  int headdim = lookup_table.size(1);
  int lutlen = lookup_table.size(2);
  assert (numheads == numheads2);

  int newveclen = newvec.size(0);
  assert (newveclen == headdim * numheads2); // for now only append one token

  int block_size = PBLOCKWIDTH;
  int num_blocks = (newveclen + PBLOCKWIDTH - 1) / PBLOCKWIDTH;

  auto options1 = torch::TensorOptions().dtype(torch::kInt32).device(torch::kCUDA);
  torch::Tensor outlier_count_per_block = torch::zeros(num_blocks,options1);

  auto options2 = torch::TensorOptions().dtype(torch::kInt32).device(torch::kCUDA);
  torch::Tensor outlier_count = torch::zeros(1,options2);

  auto options3 = torch::TensorOptions().dtype(torch::kInt32).device(torch::kCUDA);
  torch::Tensor mask = torch::zeros_like(newvec,options3);

  VecQuant4AppendVecKSparse<<<num_blocks, block_size>>>(
    mat.data_ptr<int>(),
    lookup_table.data_ptr<float>(),
    newvec.data_ptr<float>(),
    mask.data_ptr<int>(),
    outlier_count.data_ptr<int>(),
    outlier_count_per_block.data_ptr<int>(),
    outlier_threshold_lower.data_ptr<float>(),
    outlier_threshold_upper.data_ptr<float>(),
    width,
    fullwidth
  );

  torch::Tensor hostcount = outlier_count.to(torch::kCPU);
  int* count = hostcount.data_ptr<int>();
  int intcount = count[0];
  auto options4 = torch::TensorOptions().dtype(torch::kInt32).device(torch::kCUDA);
  torch::Tensor dst_indices = torch::zeros(intcount,options4);
  auto options5 = torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA);
  torch::Tensor dst_values = torch::zeros(intcount, options5);

  VecQuant4AppendVecKSparse2<<<num_blocks, block_size>>>(
    newvec.data_ptr<float>(),
    zeropoint.data_ptr<float>(),
    mask.data_ptr<int>(),
    outlier_count_per_block.data_ptr<int>(),
    dst_indices.data_ptr<int>(),
    dst_values.data_ptr<float>(),
    num_blocks
  );

  torch::Tensor row2, col2, val2, start_rows2;
  int num_threads2;

  // Deal w/ rows / cols / vals
  if (!row.numel()) {
    auto options6 = torch::TensorOptions().dtype(torch::kInt32).device(torch::kCPU);
    torch::Tensor dst_row_cpu = torch::zeros(2,options6);
    int* row_ptr = dst_row_cpu.data_ptr<int>();
    row_ptr[1] = intcount;
    torch::Tensor dst_row = dst_row_cpu.to(torch::kCUDA);

    row2 = dst_row;
    col2 = dst_indices;
    val2 = dst_values;

    // balanced part - TODO make parameterizable (currently assumes 10 nnz per thread)
    int nnz_per_thread = 10;
    int num_nonzeros = intcount;
    num_threads2 = (num_nonzeros+9) / 10 ;

    // currently initialize on CPU and copy, see if this is fast enough
    auto options8 = torch::TensorOptions().dtype(torch::kInt32).device(torch::kCUDA);
    start_rows2 = torch::full(num_threads2, kcachelen, options8);

  } else {

    auto options7 = torch::TensorOptions().dtype(torch::kInt32).device(torch::kCUDA);
    torch::Tensor dst_row_cuda = torch::full(1, col.size(0) + intcount, options7);

    int prev_num_nonzeros = col.size(0);
    int prevmax = start_rows.size(0);
    int prev_num_threads = (prev_num_nonzeros+9)/10;

    row2 = torch::cat({row,dst_row_cuda}, 0);
    if (intcount > 0) {
      col2 = torch::cat({col,dst_indices}, 0);
      val2 = torch::cat({val,dst_values}, 0);

      int nnz_per_thread = 10;
      int num_nonzeros = col2.size(0);
      num_threads2 = (num_nonzeros+9) / 10 ;
      int new_alloc = num_threads2 - prevmax;

      if (new_alloc > 0) {
        auto options9 = torch::TensorOptions().dtype(torch::kInt32).device(torch::kCUDA);
        torch::Tensor start_rows2_tmp = torch::full(new_alloc, kcachelen, options9);
        start_rows2 = torch::cat({start_rows,start_rows2_tmp}, 0);
      } else {
        start_rows2 = start_rows;
      }

    } else {
      col2 = col;
      val2 = val;
      start_rows2 = start_rows;
      int num_nonzeros = col2.size(0);
      num_threads2 = (num_nonzeros+9) / 10 ;
    }
  }

  // hack to return int
  auto options10 = torch::TensorOptions().dtype(torch::kInt32).device(torch::kCPU);
  torch::Tensor num_threads = torch::zeros(1, options10);
  int* num_threads_ptr = num_threads.data_ptr<int>();
  num_threads_ptr[0] = num_threads2;

  return {row2, col2, val2, start_rows2, num_threads, outlier_count};
}

std::vector<torch::Tensor> vecquant4appendvecVsparse_cuda(
  torch::Tensor mat,
  torch::Tensor lookup_table,
  torch::Tensor newvec,
  float zeropoint,
  torch::Tensor row,
  torch::Tensor col,
  torch::Tensor val,
  torch::Tensor start_cols,
  float outlier_threshold_lower,
  float outlier_threshold_upper,
  int vcachelen
) {

  // mat - kvcache - (num_heads, packed vseqlen, head_dim)
  int numheads = mat.size(0);
  int height = vcachelen;
  int fullheight = 8 * mat.size(1); // vseqlen
  int width = mat.size(2); // head_dim

  int newveclen = newvec.size(0);

  int block_size = PBLOCKWIDTH;
  int num_blocks = (newveclen + PBLOCKWIDTH - 1) / PBLOCKWIDTH;

  auto options1 = torch::TensorOptions().dtype(torch::kInt32).device(torch::kCUDA);
  torch::Tensor outlier_count_per_block = torch::zeros(num_blocks,options1);

  auto options2 = torch::TensorOptions().dtype(torch::kInt32).device(torch::kCUDA);
  torch::Tensor outlier_count = torch::zeros(1,options2);

  auto options3 = torch::TensorOptions().dtype(torch::kInt32).device(torch::kCUDA);
  torch::Tensor mask = torch::zeros_like(newvec,options3);

  VecQuant4AppendVecVSparse<<<num_blocks, block_size>>>(
    mat.data_ptr<int>(),
    lookup_table.data_ptr<float>(),
    newvec.data_ptr<float>(),
    mask.data_ptr<int>(),
    outlier_count.data_ptr<int>(),
    outlier_count_per_block.data_ptr<int>(),
    outlier_threshold_lower,
    outlier_threshold_upper,
    height,
    fullheight,
    width,
    numheads
  );

  torch::Tensor hostcount = outlier_count.to(torch::kCPU);
  int* count = hostcount.data_ptr<int>();
  int intcount = count[0];
  auto options4 = torch::TensorOptions().dtype(torch::kInt32).device(torch::kCUDA);
  torch::Tensor dst_indices = torch::zeros(intcount,options4);
  auto options5 = torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA);
  torch::Tensor dst_values = torch::zeros(intcount, options5);

  VecQuant4AppendVecVSparse2<<<num_blocks, block_size>>>(
    newvec.data_ptr<float>(),
    zeropoint,
    mask.data_ptr<int>(),
    outlier_count_per_block.data_ptr<int>(),
    dst_indices.data_ptr<int>(),
    dst_values.data_ptr<float>(),
    num_blocks
  );

  torch::Tensor row2, col2, val2, start_cols2;
  int num_threads2;

  // Deal w/ rows / cols / vals
  if (!col.numel()) {
    auto options6 = torch::TensorOptions().dtype(torch::kInt32).device(torch::kCPU);
    torch::Tensor dst_col_cpu = torch::zeros(2,options6);
    int* col_ptr = dst_col_cpu.data_ptr<int>();
    col_ptr[1] = intcount;
    torch::Tensor dst_col = dst_col_cpu.to(torch::kCUDA);

    row2 = dst_indices;
    col2 = dst_col;
    val2 = dst_values;

    // balanced part - TODO make parameterizable
    int nnz_per_thread = 10;
    int num_nonzeros = intcount;
    num_threads2 = (num_nonzeros+9) / 10 ;

    // currently initialize on CPU and copy, see if this is fast enough
    auto options8 = torch::TensorOptions().dtype(torch::kInt32).device(torch::kCUDA);
    start_cols2 = torch::full(num_threads2, vcachelen, options8);

  } else {
    auto options7 = torch::TensorOptions().dtype(torch::kInt32).device(torch::kCUDA);
    torch::Tensor dst_col = torch::full(1, row.size(0) + intcount, options7);

    int prev_num_nonzeros = row.size(0);
    int prevmax = start_cols.size(0);
    int prev_num_threads = (prev_num_nonzeros+9)/10;

    col2 = torch::cat({col,dst_col}, 0);
    if (intcount > 0) {
      row2 = torch::cat({row,dst_indices}, 0);
      val2 = torch::cat({val,dst_values}, 0);

      int nnz_per_thread = 10;
      int num_nonzeros = row2.size(0);
      num_threads2 = (num_nonzeros+9) / 10 ;
      int new_alloc = num_threads2 - prevmax;

      if (new_alloc > 0) {
        auto options9 = torch::TensorOptions().dtype(torch::kInt32).device(torch::kCUDA);
        torch::Tensor start_cols2_tmp = torch::full(new_alloc, vcachelen, options9);
        start_cols2 = torch::cat({start_cols,start_cols2_tmp}, 0);
      } else {
        start_cols2 = start_cols;
      }

    } else {
      row2 = row;
      val2 = val;
      start_cols2 = start_cols;
    }
  }

  // hack to return int
  auto options10 = torch::TensorOptions().dtype(torch::kInt32).device(torch::kCPU);
  torch::Tensor num_threads = torch::zeros(1, options10);
  int* num_threads_ptr = num_threads.data_ptr<int>();
  num_threads_ptr[0] = num_threads2;

  return {row2, col2, val2, start_cols2, num_threads, outlier_count};

}

void vecquant4matmul_nuq_perchannel_transposed_mha_batched_fused_cuda(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor lookup_table,
  torch::Tensor scalingfactor,
  torch::Tensor zeropoint,
  int vcachelen,
  torch::Tensor rows,
  torch::Tensor cols,
  torch::Tensor startcols,
  torch::Tensor spmat,
  int num_cols,
  int num_threads,
  int nnz
) {

  // mul - out - (score_seqlen, num_head, head_dim)
  int batch_size = mul.size(0);
  int mul_num_heads = mul.size(1);
  int mul_height = mul.size(2);
  int headdim = mul_height;

  // vec - in - (score_seqlen, num_head, vseqlen)
  int vbatch_size = vec.size(0);
  int num_vec_heads = vec.size(1);
  int vec_height = vec.size(2); // v seqlen

  // mat - kvcache - (num_heads, packed_vseqlen, head_dim)
  int numheads = mat.size(0);
  int height = vcachelen; // v sequence length
  int packedheight = height / 8;
  int fullheight = 8 * mat.size(1); // v sequence length (full max seqlen)
  int width = mat.size(2); // headdim
  assert (width == headdim);

  // lookup table - (16,)
  dim3 blocks(
    (packedheight + BLOCKHEIGHT4 - 1) / BLOCKHEIGHT4,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH,
    (numheads)
  );
  dim3 threads(BLOCKWIDTH);

  VecQuant4MatMulKernelNUQPerChannelTransposedMHABatchedFused<<<blocks, threads>>>(
    vec.data_ptr<float>(),
    mat.data_ptr<int>(),
    mul.data_ptr<float>(),
    lookup_table.data_ptr<float>(),
    scalingfactor.data_ptr<float>(),
    zeropoint.data_ptr<float>(),
    height, width, fullheight, headdim, numheads, batch_size
  );

  // balanced
  int block_size = BLOCKWIDTH;
  int num_blocks = (num_threads + BLOCKWIDTH - 1) / BLOCKWIDTH;

  SPMV_ATOMIC_CSC_BALANCED<<<num_blocks, block_size>>>(
    rows.data<int>(),
    cols.data<int>(),
    startcols.data<int>(),
    spmat.data<float>(),
    vec.data<float>(),
    mul.data<float>(),
    num_cols,
    numheads,
    vcachelen,
    headdim,
    num_threads,
    nnz
  );
}

// OPTIMIZED FUSED K KERNEL
void vecquant4matmul_nuq_perchannel_transposed_rope_mha_batched_fused_cuda(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor lookup_table,
  torch::Tensor scalingfactor,
  torch::Tensor zeropoint,
  int kcachelen,
  torch::Tensor rows,
  torch::Tensor cols,
  torch::Tensor startrows,
  torch::Tensor spmat,
  int num_rows,
  int num_threads,
  int nnz,
  float rope_theta,
  int pos_offset
) {

  // mul - out - (num_heads, qseqlen, kseqlen)
  int batch_size = mul.size(0);
  int mul_num_heads = mul.size(1);
  int mul_height = mul.size(2);

  // vec - in - (num_heads, qseqlen, head_dim)
  int vbatch_size = vec.size(0);
  int num_vec_heads = vec.size(1);
  int vec_height = vec.size(2);
  assert (vbatch_size == batch_size);

  // mat - kvcache - (num_heads, head_dim, kseqlen)
  int numheads = mat.size(0);
  int height = mat.size(1); // headdim
  int width = kcachelen; // sequence length
  int fullwidth = mat.size(2); // max sequence length
  assert(width == mul_height);

  int headdim = 8 * height;

  dim3 blocks(
    (height + BLOCKHEIGHT4 - 1) / BLOCKHEIGHT4,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH,
    (numheads)
  );
  dim3 threads(BLOCKWIDTH);


  VecQuant4MatMulKernelNUQPerChannelTransposedRopeMHABatchedFused<<<blocks, threads>>>(
    vec.data_ptr<float>(),
    mat.data_ptr<int>(),
    mul.data_ptr<float>(),
    lookup_table.data_ptr<float>(),
    scalingfactor.data_ptr<float>(),
    zeropoint.data_ptr<float>(),
    height,
    width,
    fullwidth,
    headdim,
    numheads,
    batch_size,
    rope_theta,
    pos_offset
  );

  // check if no nonzeros yet
  if (num_threads > 0) {

    // TODO: need to make this support batching for sparse kernel
    int block_size = BLOCKWIDTH;
    int num_blocks = (num_threads + BLOCKWIDTH - 1) / BLOCKWIDTH;

    SPMV_ATOMIC_CSR_ROPE_BALANCED<<<num_blocks, block_size>>>(
      rows.data<int>(),
      cols.data<int>(),
      startrows.data<int>(),
      spmat.data<float>(),
      vec.data<float>(),
      mul.data<float>(),
      num_rows,
      numheads,
      kcachelen,
      headdim,
      num_threads,
      nnz,
      rope_theta,
      pos_offset
    );

  }
}

template <typename scalar_t>
__global__ void SPMV_ATOMIC_CSR_ROPE_BALANCED(
    const       int* __restrict__ rows,
    const       int* __restrict__ cols,
    const       int* __restrict__ startrows,
    const  scalar_t* __restrict__ mat,
    const  scalar_t* __restrict__ vec,
           scalar_t* __restrict__ mul,
    const  int num_rows,
    int numheads,
    int seqlen,
    int headdim,
    int num_threads,
    int nnz,
    float rope_theta,
    int pos_offset
) {

    int nnz_per_thread = (nnz + num_threads - 1) / num_threads;
    int threadid = blockIdx.x * blockDim.x + threadIdx.x;

    if (threadid < num_threads) {

      int row = startrows[threadid];
      int nextrow = -1;
      if (row != -1) {
        nextrow = rows[row+1];

        // extra check to make sure we don't start with an empty row!
        while (nextrow == threadid * nnz_per_thread) {
          row += 1;
          if (row < num_rows) {
            nextrow = rows[row+1];
          } else {
            break;
          }
        }
      }

      float theta;
      float sign;
      float c, s;

      if (threadid*nnz_per_thread < nnz && row != -1) {

          int max = (threadid+1)*nnz_per_thread;
          if (nnz < max) {
            max = nnz;
          }

          float dot = 0;

          for (int i = threadid * nnz_per_thread; i < max; i++) {

              int col = cols[i];
              float mat_tmp = mat[i];

              int headid = col / headdim;
              int channel_head_off = col % headdim; // needed for RoPE pos

              // RoPE embeddings
              theta = powf ( rope_theta , (-2 * __int2float_rd(channel_head_off % (headdim/2)) / headdim) );
              sign = (channel_head_off < (headdim/2)) ? 1 : -1;
              c = cosf(theta * (row + pos_offset));
              s = sinf(theta * (row + pos_offset));

              // compute dot products
              int col2 = ((channel_head_off + (headdim/2)) % headdim ) + headid * headdim;
              dot = mat_tmp * c * vec[col];
              dot += sign * mat_tmp * s * vec[col2];

              atomicAdd(&mul[headid * seqlen + row], dot);

              if (i + 1 == nextrow) { // finish & move on to next row

                  dot = 0;

                  while (i + 1 == nextrow) { // while loop is to deal with cases where there are entire zero rows
                      row += 1;
                      if (row < num_rows) {
                          nextrow = rows[row+1];
                      } else {
                          nextrow = -1;
                          break;
                      }
                  }
              }

          }
      }
    }
}

template <typename scalar_t>
__global__ void SPMV_ATOMIC_CSC_BALANCED(
    const       int* __restrict__ rows,
    const       int* __restrict__ cols,
    const       int* __restrict__ startcols,
    const  scalar_t* __restrict__ mat,
    const  scalar_t* __restrict__ vec,
           scalar_t* __restrict__ mul,
    const  int num_cols,
    int numheads,
    int seqlen,
    int headdim,
    int num_threads,
    int nnz
) {

    int nnz_per_thread = (nnz + num_threads - 1) / num_threads;
    int threadid = blockIdx.x * blockDim.x + threadIdx.x;

    if (threadid < num_threads) {

      int col = startcols[threadid];
      int nextcol = -1;
      if (col != -1) {
        nextcol = cols[col+1];

        // extra check to make sure we don't start with an empty row!
        while (nextcol == threadid * nnz_per_thread) {
          col += 1;
          if (col < num_cols) {
            nextcol = cols[col+1];
          } else {
            break;
          }
        }
      }


      if (threadid*nnz_per_thread < nnz && col != -1) {

          int max = (threadid+1)*nnz_per_thread;
          if (nnz < max) {
            max = nnz;
          }

          float dot = 0;

          for (int i = threadid * nnz_per_thread; i < max; i++) {

              int row = rows[i];
              int headid = row / headdim;
              float vval = vec[headid * seqlen + col];

              dot = mat[i] * vval;

              atomicAdd(&mul[row], dot);

              if (i + 1 == nextcol) { // finish & move on to next row
                  dot = 0;

                  while (i + 1 == nextcol) { // while loop is to deal with cases where there are entire zero rows
                      col += 1;
                      if (col < num_cols) {
                          nextcol = cols[col+1];
                      } else {
                          nextcol = -1;
                          break;
                      }
                  }
              }
          }
      }
    }
}

__global__ void VecQuant4AppendVecKSparse(
           int* __restrict__ mat,
  const  float* __restrict__ lookup_table,
  const  float* __restrict__ newvec,
  int*  __restrict__ mask,
  int*  __restrict__ outlier_count,
  int*  __restrict__ outlier_count_per_block,
  const float* __restrict__ outlier_threshold_lower,
  const float* __restrict__ outlier_threshold_upper,
  int width,
  int fullwidth
) {

  int packedoffset = PBLOCKHEIGHT4 * blockIdx.x;

  //Modified dequant block
  __shared__ float deq2[16][PBLOCKWIDTH];
  int off = threadIdx.x;

  int lut_row = PBLOCKWIDTH * blockIdx.x + threadIdx.x;
  int row_offset = lut_row * 16;

  // get value of vec to pack
  int offset = PBLOCKWIDTH * blockIdx.x + threadIdx.x;
  float newvecval = newvec[offset];

  // loop over LUT to find smallest entry
  for (int val = 0; val < 16; val += 1) {
    int lut_index = row_offset + val;
    deq2[val][off] = fabsf(lookup_table[lut_index] - newvecval);
  }

  // check for outliers before packing
  float lower_threshold = outlier_threshold_lower[lut_row];
  float upper_threshold = outlier_threshold_upper[lut_row];

  int num_outliers = 0;

  int smallest_idx = 0;
  if ((newvecval < lower_threshold) || (newvecval > upper_threshold)) {
    smallest_idx = 7; //zero-point
    mask[lut_row] = 1; // set boolean mask
    num_outliers += 1;
  } else {
    // find index of smallest entry in lut
    float prev_val = deq2[0][off];

    for (int val = 1; val < 16; val += 1) {
      if (deq2[val][off] < prev_val) {
        prev_val = deq2[val][off];
        smallest_idx = val;
      }
    }
  }

  // update mat entry using computed idx
  int row = packedoffset + (threadIdx.x / 8);
  int i = fullwidth * row + width;
  int word_offset = (threadIdx.x % 8) * 4;
  int word_to_add = (smallest_idx << word_offset);
  atomicAdd(&mat[i], word_to_add);
  atomicAdd(&outlier_count[0], num_outliers);
  atomicAdd(&outlier_count_per_block[blockIdx.x], num_outliers);
}

__global__ void VecQuant4AppendVecKSparse2(
  const  float* __restrict__ newvec,
  float* __restrict__ zeropoint,
  int*  __restrict__ mask,
  int*  __restrict__ outlier_count_per_block,
  int*  __restrict__ dst_indices,
  float*  __restrict__ dst_values,
  int num_blocks
) {
  __shared__ int limits[2];
  if (threadIdx.x == 0) {
      int outlier_offset = 0;
      for (int i=0; i<blockIdx.x; i++) {
        outlier_offset += outlier_count_per_block[i];
      }
      limits[0] = outlier_offset;
      limits[1] = outlier_offset + outlier_count_per_block[blockIdx.x];
  }
  __syncthreads();

  if (threadIdx.x == 0) {
    int insert_loc = limits[0];
    for (int i=0; i<PBLOCKWIDTH; i++) {
      int outlier_offset = PBLOCKWIDTH * blockIdx.x + i;
      if (mask[outlier_offset] != 0) {
        dst_indices[insert_loc] = outlier_offset;
        float newvecval = newvec[outlier_offset];
        float zeropointval = zeropoint[outlier_offset];
        float insert_val = newvecval - zeropointval;
        dst_values[insert_loc] = insert_val;
        insert_loc += 1;
      }
    }
  }
}

__global__ void VecQuant4AppendVecVSparse(
           int* __restrict__ mat,
         float* __restrict__ lookup_table,
  const  float* __restrict__ newvec,
  int* __restrict__ mask,
  int* __restrict__ outlier_count,
  int*  __restrict__ outlier_count_per_block,
  float outlier_threshold_lower,
  float outlier_threshold_upper,
  int height,
  int fullheight,
  int width,
  int numheads
) {

  // offset across heads
  int packedheadheight = (fullheight * PBLOCKHEIGHT4) / PBLOCKWIDTH;
  int headid = (blockIdx.x / (128 / PBLOCKWIDTH)); // TODO: only works for 7B
  int packedheadoffset = headid * packedheadheight;

  // within a head
  int packedoffset = (height * PBLOCKHEIGHT4) / PBLOCKWIDTH;
  int packedmod = height % 8;

  // Modified dequant block -
  __shared__ float deq2[16][PBLOCKWIDTH];
  int off = threadIdx.x;

  // get value of vec to pack
  int offset = PBLOCKWIDTH * blockIdx.x + threadIdx.x;
  float newvecval = newvec[offset];

  // sf / zpt
  float scalingfactor = (outlier_threshold_upper - outlier_threshold_lower) / 2;
  float zeropoint = (outlier_threshold_upper + outlier_threshold_lower) / 2;

  // loop over LUT to find smallest entry
  for (int val = 0; val < 16; val += 1) {
    float lutval = lookup_table[val];
    deq2[val][off] = fabsf((lutval * scalingfactor + zeropoint) - newvecval);
  }

  // check for outliers before packing
  int num_outliers = 0;
  int smallest_idx = 0;
  if ((newvecval < outlier_threshold_lower) || (newvecval > outlier_threshold_upper)) {
    smallest_idx = 7; //zero-point
    mask[offset] = 1; // set boolean mask
    num_outliers += 1;
  } else {
    // find index of smallest entry in lut
    float prev_val = deq2[0][off];
    for (int val = 1; val < 16; val += 1) {
      if (deq2[val][off] < prev_val) {
        prev_val = deq2[val][off];
        smallest_idx = val;
      }
    }
  }

  // update mat entry using computed idx
  int row = packedoffset + packedheadoffset;
  int i = width * row + (offset % 128); // TODO make more general
  int word_offset = packedmod * 4;
  int word_to_add = (smallest_idx << word_offset);
  atomicAdd(&mat[i], word_to_add);
  atomicAdd(&outlier_count[0], num_outliers);
  atomicAdd(&outlier_count_per_block[blockIdx.x], num_outliers);
}

__global__ void VecQuant4AppendVecVSparse2(
  const  float* __restrict__ newvec,
  float zeropoint,
  int*  __restrict__ mask,
  int*  __restrict__ outlier_count_per_block,
  int*  __restrict__ dst_indices,
  float*  __restrict__ dst_values,
  int num_blocks
) {
  __shared__ int limits[2];
  if (threadIdx.x == 0) {
      int outlier_offset = 0;
      for (int i=0; i<blockIdx.x; i++) {
        outlier_offset += outlier_count_per_block[i];
      }
      limits[0] = outlier_offset;
      limits[1] = outlier_offset + outlier_count_per_block[blockIdx.x];
  }
  __syncthreads();

  if (threadIdx.x == 0) {
    int insert_loc = limits[0];
    for (int i=0; i<PBLOCKWIDTH; i++) {
      int outlier_offset = PBLOCKWIDTH * blockIdx.x + i;
      if (mask[outlier_offset] != 0) {
        dst_indices[insert_loc] = outlier_offset;
        float newvecval = newvec[outlier_offset];
        float insert_val = newvecval - zeropoint;
        dst_values[insert_loc] = insert_val;
        insert_loc += 1;
      }
    }
  }
}

__global__ void VecQuant4MatMulKernelNUQPerChannelTransposedMHABatchedFused(
    const  float* __restrict__ vec,
    const    int* __restrict__ mat,
           float* __restrict__ mul,
    const  float* __restrict__ lookup_table,
    const  float* __restrict__ scalingfactor,
    const  float* __restrict__ zeropoint,
    int height,
    int width,
    int fullheight,
    int headdim,
    int numheads,
    int batch_size
) {

  int headid = blockIdx.z;
  int headoffset = width * headid;
  int sloffset = fullheight * headid; // in terms of number of logical rows
  int packedsloffset = (sloffset * BLOCKHEIGHT4) / BLOCKWIDTH; // in terms of packed words

  int row = packedsloffset + BLOCKHEIGHT4 * blockIdx.x;
  int col = BLOCKWIDTH * blockIdx.y + threadIdx.x;

  __shared__ float blockvec[BLOCKWIDTH];
  __shared__ float sf[BLOCKWIDTH];
  __shared__ float zpt[BLOCKWIDTH];
  __shared__ float curr_zpt;
  __shared__ float curr_sf;

  //Modified dequant block
  __shared__ float deq2[16][BLOCKWIDTH];
  int off = threadIdx.x;

  // CHECK 1 for sequence length
  for (int val = 0; val < 16; val += 1) {
    deq2[val][off] = lookup_table[val];
  }

  int i;
  int k = 0;
  float res = 0;

  unsigned int tmp;
  i = width * row + col;
  k = 0;

  int logical_row = BLOCKWIDTH * blockIdx.x + threadIdx.x; // don't use fullheight here for vec
  blockvec[threadIdx.x] = vec[height * headid + logical_row];
  sf[threadIdx.x] = scalingfactor[logical_row];
  zpt[threadIdx.x] = zeropoint[logical_row];

  __syncthreads();

  // TODO: not needed for benchmarking, but add check
  // incase sl is not a multiple of 128

  while (k < BLOCKWIDTH) {
    tmp = as_unsigned(mat[i]);

    curr_zpt = zpt[k];
    curr_sf = sf[k];
    res += (deq2[(tmp >> 0) & 0xf][off] * curr_sf + curr_zpt) * blockvec[k];
    k += 1;
    curr_zpt = zpt[k];
    curr_sf = sf[k];
    res += (deq2[(tmp >> 4) & 0xf][off] * curr_sf + curr_zpt) * blockvec[k];
    k += 1;
    curr_zpt = zpt[k];
    curr_sf = sf[k];
    res += (deq2[(tmp >> 8) & 0xf][off] * curr_sf + curr_zpt) * blockvec[k];
    k += 1;
    curr_zpt = zpt[k];
    curr_sf = sf[k];
    res += (deq2[(tmp >> 12) & 0xf][off] * curr_sf + curr_zpt) * blockvec[k];
    k += 1;
    curr_zpt = zpt[k];
    curr_sf = sf[k];
    res += (deq2[(tmp >> 16) & 0xf][off] * curr_sf + curr_zpt) * blockvec[k];
    k += 1;
    curr_zpt = zpt[k];
    curr_sf = sf[k];
    res += (deq2[(tmp >> 20) & 0xf][off] * curr_sf + curr_zpt) * blockvec[k];
    k += 1;
    curr_zpt = zpt[k];
    curr_sf = sf[k];
    res += (deq2[(tmp >> 24) & 0xf][off] * curr_sf + curr_zpt) * blockvec[k];
    k += 1;
    curr_zpt = zpt[k];
    curr_sf = sf[k];
    res += (deq2[(tmp >> 28) & 0xf][off] * curr_sf + curr_zpt) * blockvec[k];
    k += 1;

    i += width;
  }

  atomicAdd(&mul[headoffset + col], res);
}

__global__ void VecQuant4MatMulKernelNUQPerChannelTransposedRopeMHABatchedFused(
    const  float* __restrict__ vec,
    const    int* __restrict__ mat,
           float* __restrict__ mul,
    const  float* __restrict__ lookup_table,
    const  float* __restrict__ scalingfactor,
    const  float* __restrict__ zeropoint,
    int height,
    int width,
    int fullwidth,
    int headdim,
    int numheads,
    int batch_size,
    float rope_theta,
    int pos_offset
) {

  int headid = blockIdx.z;
  int headoffset = headdim * headid; // in terms of number of logical rows

  int packedheadoffset = (headoffset * BLOCKHEIGHT4) / BLOCKWIDTH; // in terms of packed words

  int row = packedheadoffset + BLOCKHEIGHT4 * blockIdx.x;
  int col = BLOCKWIDTH * blockIdx.y + threadIdx.x;

  __shared__ float blockvec[BLOCKWIDTH];
  __shared__ float blockvec2[BLOCKWIDTH];

  __shared__ float sf[BLOCKWIDTH];
  __shared__ float zpt[BLOCKWIDTH];
  __shared__ float thetavec[BLOCKWIDTH];

  //Modified dequant block
  __shared__ float deq2[16][BLOCKWIDTH];
  int off = threadIdx.x;

  // CHECK 1 for sequence length
  for (int val = 0; val < 16; val += 1) {
    deq2[val][off] = lookup_table[val];
  }

  int headdim2 = headdim/2;

  sf[threadIdx.x] = scalingfactor[(row / BLOCKHEIGHT4) * BLOCKWIDTH + threadIdx.x];
  zpt[threadIdx.x] = zeropoint[(row / BLOCKHEIGHT4) * BLOCKWIDTH + threadIdx.x];
  thetavec[threadIdx.x] = powf ( rope_theta , (-2 * __int2float_rd(threadIdx.x % headdim2) / __int2float_rd(headdim)) );

  __syncthreads();

  int i;
  int k = 0;
  float res = 0;

  unsigned int tmp;

  // for RoPE
  int pos = col + pos_offset;
  float tmp1, tmp2;
  float c, s;
  int k2;
  float tmp3;
  float theta = 0;
  float sign;
  float angle;

  float subtract_pi = 0;

  for (int b = 0; b < batch_size; b++) {

    __syncthreads();
    i = fullwidth * row + col;

    int vec_batch_offset = b * headdim * numheads;
    int headdim2 = headdim/2;
    blockvec[threadIdx.x] = vec[vec_batch_offset + (row / BLOCKHEIGHT4) * BLOCKWIDTH + threadIdx.x];
    blockvec2[threadIdx.x] = vec[vec_batch_offset + (row / BLOCKHEIGHT4) * BLOCKWIDTH + (threadIdx.x+headdim2)%headdim];

    __syncthreads();

    k = 0;
    res = 0;

    // CHECK 1 for sequence length
    if (col < width) {

      while (k < BLOCKWIDTH) {
        tmp = as_unsigned(mat[i]);
        sign = (k<64) ? 1 : -1; // wouldn't work for cyclic

        tmp1 = deq2[(tmp >>  0) & 0xf][off] * sf[k] + zpt[k];
        theta = thetavec[k];
        angle = theta * pos;
        __sincosf(angle, &s, &c); // sincosf(theta * pos, &s, &c);

        res += tmp1 * c * blockvec[k];
        res += sign * tmp1 * s * blockvec2[k];
        k += 1;

        tmp1 = deq2[(tmp >>  4) & 0xf][off] * sf[k] + zpt[k];
        theta = thetavec[k];
        angle = theta * pos;
        __sincosf(angle, &s, &c); // sincosf(theta * pos, &s, &c);

        res += tmp1 * c * blockvec[k];
        res += sign * tmp1 * s * blockvec2[k];
        k += 1;

        tmp1 = deq2[(tmp >>  8) & 0xf][off] * sf[k] + zpt[k];
        theta = thetavec[k];
        angle = theta * pos;
        __sincosf(angle, &s, &c); // sincosf(theta * pos, &s, &c);

        res += tmp1 * c * blockvec[k];
        res += sign * tmp1 * s * blockvec2[k];
        k += 1;

        tmp1 = deq2[(tmp >>  12) & 0xf][off] * sf[k] + zpt[k];
        theta = thetavec[k];
        angle = theta * pos;
        __sincosf(angle, &s, &c); // sincosf(theta * pos, &s, &c);

        res += tmp1 * c * blockvec[k];
        res += sign * tmp1 * s * blockvec2[k];
        k += 1;

        tmp1 = deq2[(tmp >>  16) & 0xf][off] * sf[k] + zpt[k];
        theta = thetavec[k];
        angle = theta * pos;
        __sincosf(angle, &s, &c); // sincosf(theta * pos, &s, &c);

        res += tmp1 * c * blockvec[k];
        res += sign * tmp1 * s * blockvec2[k];
        k += 1;

        tmp1 = deq2[(tmp >>  20) & 0xf][off] * sf[k] + zpt[k];
        theta = thetavec[k];
        angle = theta * pos;
        __sincosf(angle, &s, &c); // sincosf(theta * pos, &s, &c);

        res += tmp1 * c * blockvec[k];
        res += sign * tmp1 * s * blockvec2[k];
        k += 1;

        tmp1 = deq2[(tmp >>  24) & 0xf][off] * sf[k] + zpt[k];
        theta = thetavec[k];

        angle = theta * pos;
        __sincosf(angle, &s, &c); // sincosf(theta * pos, &s, &c);

        res += tmp1 * c * blockvec[k];
        res += sign * tmp1 * s * blockvec2[k];
        k += 1;

        tmp1 = deq2[(tmp >>  28) & 0xf][off] * sf[k] + zpt[k];
        theta = thetavec[k];

        angle = theta * pos;
        __sincosf(angle, &s, &c); // sincosf(theta * pos, &s, &c);

        res += tmp1 * c * blockvec[k];
        res += sign * tmp1 * s * blockvec2[k];
        k += 1;

        i += fullwidth;
      }

      int mul_batch_offset = b * width * numheads;
      atomicAdd(&mul[mul_batch_offset + headid * width + col], res);
    }
  }
}
